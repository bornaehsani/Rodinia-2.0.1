#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/types.h>
#include <unistd.h>
#include <errno.h>
#include <sys/time.h>

#include <hip/hip_vector_types.h>

// includes, kernels
#include "common.h"
#include "suffix-tree.h"

#include "mummergpu.h"

int USE_PRINT_KERNEL = 1;

#define BREATHING_ROOM (16 * 1024 * 1024)
#define BASES_PER_TREE_PAGE 8388608
//#define BASES_PER_TREE_PAGE 7000000
#define BLOCKSIZE 256
unsigned int cuda_calls = 0;
void trap_dbg() {
  fprintf(stderr, "Trapped\n");
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold(MatchResults* results,
                 char* refstr,
                 char* queries,
                 int* queryAddrs,
                 int* queryLengths,
                 PixelOfNode* nodeTexture,
                 PixelOfChildren* childrenTexture,
                 int numQueries,
                 int mismatch_length,
                 int rc);

extern "C"
void getReferenceString(const char * filename, char** refstr, size_t* reflen);

extern "C"
void createTreeTexture(const char * filename,
                       PixelOfNode** nodeTexture,
                       PixelOfChildren** childrenTexture,
                       unsigned int* width,
                       unsigned int* node_height,
                       unsigned int* children_height,
                       AuxiliaryNodeData** aux_data,
                       int* num_match_coords,
                       int min_match_len,
                       Statistics* statistics,
                       const char * dotfilename,
                       const char * texfilename);

extern "C"
void getQueriesTexture(int qfile,
                       char** queryTexture,
                       size_t* queryLength,
                       int** queryAddrs,
                       char*** queryNames,
                       int** queryLengths,
                       unsigned int* numQueries,
                       unsigned int* num_match_coords,
                       unsigned int device_memory_avail,
                       int min_match_length,
                       bool rc);

/*
extern "C"
int lookupNumLeaves(ReferencePage * page, TextureAddress addr);
*/

void printAlignments(ReferencePage* page,
                     Alignment* alignments,
                     char* query,
                     int qrylen,
                     TextureAddress nodeid,
                     int qrypos,
                     int edge_depth,
                     int min_match,
                     bool rc,
                     bool forwardcoordinates);

int  countLeafNodes(int nodeid);

extern "C"
void mapQueriesEndToEnd(MatchContext* ctx,
                        ReferencePage* page,
                        MatchInfo* h_matches,
                        unsigned int numMatches,
                        Alignment* h_alignments,
                        unsigned int numAligments);

char *  createTimer() {
  unsigned int * ptr = (unsigned int *) malloc(sizeof(struct Timer_t));
  memset(ptr, 0, sizeof(struct Timer_t));
  return (char *) ptr;
}

void startTimer(char * ptr) {
  gettimeofday(&(((struct Timer_t *)ptr)->start_m), NULL);
}

void stopTimer(char * ptr) {
  gettimeofday(&(((struct Timer_t *)ptr)->end_m), NULL);
}

float getTimerValue(char * ptr) {
  Timer_t * timer = (Timer_t*) ptr;

  if (timer == NULL) {
    fprintf(stderr, "Uninitialized timer!!!\n");
    return 0.0;
  }

  if (timer->end_m.tv_sec == 0) {
    stopTimer(ptr);
  }

  return  (float) (1000.0 * (timer->end_m.tv_sec - timer->start_m.tv_sec)
                   + (0.001 *  (timer->end_m.tv_usec - timer->start_m.tv_usec)));
}

void deleteTimer(char * ptr) {
  free((Timer_t *)ptr);
}

extern "C"
int createReference(const char* fromFile, Reference* ref) {
  if (!fromFile || !ref)
    return -1;

  char * loadreftimer = createTimer();
  startTimer(loadreftimer);

  getReferenceString(fromFile, &(ref->str), &(ref->len));

  stopTimer(loadreftimer);
  ref->t_load_from_disk += getTimerValue(loadreftimer);
  deleteTimer(loadreftimer);

  return 0;
}

extern "C"
int destroyReference(Reference* ref) {
  free(ref->h_node_tex_array);
  free(ref->h_children_tex_array);
  free(ref->str);
#if REORDER_REF
  free(ref->h_ref_array);
#endif

  free(ref->aux_data);
#if TREE_ACCESS_HISTOGRAM
  free(ref->h_node_hist);
  free(ref->h_child_hist);
#endif
  ref->str = NULL;
  ref->len = 0;

  return 0;
}

extern "C"
int createQuerySet(const char* fromFile, QuerySet* queries) {

  fprintf(stderr, "Opening %s...\n", fromFile);
  int qfile = open(fromFile, O_RDONLY);

  if (qfile == -1) {
    fprintf(stderr, "Can't open %s: %d\n", fromFile, errno);
    exit (1);
  }

  queries->qfile = qfile;

  return 0;
}

extern "C"
int destroyQuerySet(QuerySet* queries) {

  if (queries->qfile)
    close(queries->qfile);

  return 0;
}

extern "C"
void printStringForError(int err) {

}

extern "C"
int createMatchContext(Reference* ref,
                       QuerySet* queries,
                       MatchResults* matches,
                       bool on_cpu,
                       int min_match_length,
                       char* stats_file,
                       bool reverse,
                       bool forwardreverse,
                       bool forwardcoordinates,
                       bool showQueryLength,
                       char* dotfilename,
                       char* texfilename,
                       MatchContext* ctx) {

  ctx->queries = queries;
  ctx->ref = ref;
  ctx->full_ref = ref->str;
  ctx->full_ref_len = ref->len;

  ctx->on_cpu = on_cpu;
  ctx->min_match_length = min_match_length;
  ctx->stats_file = stats_file;
  ctx->reverse = reverse;
  ctx->forwardreverse = forwardreverse;
  ctx->forwardcoordinates = forwardcoordinates;
  ctx->show_query_length = showQueryLength;
  ctx->dotfilename = dotfilename;
  ctx->texfilename = texfilename;
  return 0;
}


extern "C"
int destroyMatchContext(MatchContext* ctx) {
  free(ctx->full_ref);
  //destroyReference(ctx->ref);
  destroyQuerySet(ctx->queries);
  return 0;
}

void buildReferenceTexture(Reference* ref,
                           char* full_ref,
                           size_t begin,
                           size_t end,
                           int min_match_len,
                           char* dotfilename,
                           char* texfilename,
                           Statistics* statistics) {
  fprintf(stderr, "Building reference texture...\n");

  PixelOfNode* nodeTexture = NULL;
  PixelOfChildren * childrenTexture = NULL;

  unsigned int width = 0;
  unsigned int node_height = 0;
  unsigned int children_height = 0;

  AuxiliaryNodeData* aux_data = NULL;
  int num_nodes;

  char * loadreftimer = createTimer();
  startTimer(loadreftimer);

  ref->len = end - begin + 3;
  ref->str = (char*)malloc(ref->len);
  ref->str[0] = 's';
  strncpy(ref->str + 1, full_ref + begin, ref->len - 3);
  strcpy(ref->str + ref->len - 2, "$");

  stopTimer(loadreftimer);
  statistics->t_ref_from_disk += getTimerValue(loadreftimer) + ref->t_load_from_disk;
  deleteTimer(loadreftimer);

  createTreeTexture(ref->str,
                    &nodeTexture,
                    &childrenTexture,
                    &width,
                    &node_height,
                    &children_height,
                    &aux_data,
                    &num_nodes,
                    min_match_len,
                    statistics,
                    dotfilename,
                    texfilename);

  ref->h_node_tex_array = nodeTexture;
  ref->h_children_tex_array = childrenTexture;
  ref->tex_width = width;
  ref->tex_node_height = node_height;
  ref->tex_children_height = children_height;

#if TREE_ACCESS_HISTOGRAM
  ref->h_node_hist = (int*)calloc(width * node_height, sizeof(int));
  ref->h_child_hist = (int*)calloc(width * children_height, sizeof(int));
#endif

  ref->aux_data = aux_data;
  ref->num_nodes = num_nodes;

  ref->bytes_on_board = (width * node_height * sizeof(PixelOfNode)) +
                        (width * children_height * sizeof(PixelOfChildren));
  fprintf(stderr, "This tree will need %ld bytes on the board\n", ref->bytes_on_board);

#if REORDER_REF
  char * reordertimer = createTimer();
  startTimer(reordertimer);

  unsigned int refpitch = ref->pitch = 65536;
  int numrows = ceil(ref->len / ((float)refpitch));
  int blocksize = 4;
  numrows += blocksize;

  int refstrsize = numrows * refpitch;
  ref->h_ref_array = (char *) malloc(refstrsize);
  ref->bytes_on_board += refstrsize;

  fprintf(stderr, "The refstr (reordered) requires %d bytes\n", refstrsize);

  int z_max = numrows * refpitch;
  for (int z = 0; z < z_max; z++) {
    ref->h_ref_array[z] = 'Z';
  }

  int x, y;
  int maxx = 0, maxy = 0;

  size_t reflen = ref->len;
  char* refstr = ref->str;


  int block_dim = refpitch * blocksize;
  for (int i = 0; i < reflen; i++) {
    int bigx = i % (block_dim); // ref string reorder
    int bigy = i / (block_dim);

    y = bigy * blocksize + bigx % blocksize;
    x = bigx / blocksize;

    //   printf("%d: (%d,%d)=%c\n", i, x, y, refstr[i]);

    assert(x < refpitch);
    assert(y < numrows);

    ref->h_ref_array[y*refpitch+x] = refstr[i];

    if (x > maxx) {
      maxx = x;
    }
    if (y > maxy) {
      maxy = y;
    }
  }

  if ((maxx >= refpitch) || (maxy >= numrows)) {
    fprintf(stderr, "ERROR: maxx: %d refpitch: %d, maxy: %d numrows: %d\n",
            maxx,    refpitch,     maxy,    numrows);

    exit(1);
  }
  stopTimer(reordertimer);
  if (statistics)
    statistics->t_reorder_ref_str += getTimerValue(reordertimer);
  deleteTimer(reordertimer);
#else
  fprintf(stderr, "The refstr requires %ld bytes\n", ref->len);
  ref->bytes_on_board += ref->len;
#endif


}

void loadReferenceTexture(MatchContext* ctx) {
  Reference* ref = ctx->ref;
  int numrows = ceil(ref->len / ((float)ref->pitch));
  int blocksize = 4;
  numrows += blocksize;

  hipChannelFormatDesc refTextureDesc =
    hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindSigned);

  if (!ctx->on_cpu) {

  } else {
    ref->d_ref_array = NULL;
  }
}


void unloadReferenceString(Reference* ref) {
  ref->d_ref_array = NULL;
}


//loads a tree and text for [begin, end) in the reference
void loadReference(MatchContext* ctx) {

  Reference* ref = ctx->ref;

  ref->bytes_on_board = 0;

  loadReferenceTexture(ctx);

  if (!ctx->on_cpu) {

  } else {
    ref->d_node_tex_array = NULL;
    ref->d_children_tex_array = NULL;
  }
}



void dumpQueryBlockInfo(QuerySet* queries) {
  fprintf(stderr, "\tProcessing queries %s to %s\n",
          queries->h_names[0],
          queries->h_names[queries->count-1]);
}

void loadQueries(MatchContext* ctx) {
  QuerySet* queries = ctx->queries;
  queries->bytes_on_board = 0;

  unsigned int numQueries = queries->count;

  if (!ctx->on_cpu) {

  } else {
    queries->d_addrs_tex_array = NULL;
    queries->d_tex_array = NULL;
    queries->d_lengths_array = NULL;
    fprintf(stderr, " allocated %ld bytes\n", 2 * numQueries*sizeof(int) + queries->texlen);
  }


}


// Computes the location of the first MatchCoord for a given query.  NOTE:
// Do NOT use this function if COALESCED_QUERIES == 1
inline int match_coord_addrs(int qryid, int qry_addrs, int match_length) {
  return qry_addrs - qryid * (match_length + 1);
}

// Construct the offset table for a set of queries.  This table will be used
// by the printing functions, and if COALESCED_QUERIES == 1, by the matching
// kernel.
void buildCoordOffsetArray(MatchContext* ctx,
                           int** h_coord_offset_array,
                           unsigned int* num_coords) {
  int numCoords = 0;
  int match_length = ctx->min_match_length;
  int numQueries = ctx->queries->count;
  int* lengths = ctx->queries->h_lengths_array;

  int* coord_offsets = (int*)calloc(numQueries, sizeof(int));

#if COALESCED_QUERIES

  for (unsigned int i = 0; i < numQueries; i += WARP_SIZE) {
    // Every query in this warp will need at least this many coords
    int max_num_coords = 0;
    for (unsigned int j = 0; j < WARP_SIZE && (i + j) < numQueries; ++j) {
      int num_coords = lengths[i + j] - match_length + 1;
      if ( max_num_coords <  num_coords)
        max_num_coords = num_coords;
    }

    unsigned int block_size = max_num_coords * WARP_SIZE;

    for (unsigned int j = 0; j < WARP_SIZE && (i + j) < numQueries; ++j) {
      ctx->results.h_coord_tex_array[i + j] = numCoords + j;
    }
    numCoords += block_size;
  }
#else
  for (unsigned int i = 0; i < numQueries; ++i) {
    int qryoffset = ctx->queries->h_addrs_tex_array[i];
    coord_offsets[i] = match_coord_addrs(i, qryoffset, match_length);
  }
  if (numQueries > 0) {
    unsigned int last_qry = numQueries - 1;
    unsigned int last_qry_len = lengths[last_qry] - match_length + 1;
    numCoords = coord_offsets[last_qry] + last_qry_len;
    fprintf(stderr, "Need %d match coords for this result array\n",
            numCoords);
  }
#endif
  *num_coords = numCoords;
  *h_coord_offset_array = coord_offsets;
}


void loadResultBuffer(MatchContext* ctx) {
  unsigned int numQueries = ctx->queries->count;

  assert (numQueries);

  char* offsettimer = createTimer();
  startTimer(offsettimer);

  buildCoordOffsetArray(ctx,
                        &(ctx->results.h_coord_tex_array),
                        &(ctx->results.numCoords));

  stopTimer(offsettimer);
  ctx->statistics.t_build_coord_offsets += getTimerValue(offsettimer);
  deleteTimer(offsettimer);

  unsigned int numCoords = ctx->results.numCoords;
  fprintf(stderr, "Allocating result array for %d queries (%ld bytes) ...",
          numQueries, numCoords*sizeof(MatchCoord) );

  size_t boardFreeMemory = 0;
  size_t total_mem = 0;

  fprintf(stderr,"board free memory: %ld total memory: %ld\n",
          boardFreeMemory, total_mem);

  ctx->results.h_match_coords = (MatchCoord*)calloc( numCoords, sizeof(MatchCoord));
  /* pinned memory did not improve runtime
  hipHostMalloc(&ctx->results.h_match_coords, numCoords * sizeof(MatchCoord));
  memset(ctx->results.h_match_coords, 0, numCoords * sizeof(MatchCoord));
  */
  if (ctx->results.h_match_coords == NULL) {
    trap_dbg();
    exit(EXIT_FAILURE);
  }

  if (!ctx->on_cpu) {

  } else {
    ctx->results.d_match_coords = NULL;
  }

  fprintf(stderr, "done\n");
}




int flushOutput();
int addToBuffer(char* string);

char numbuffer[32];

MatchCoord* coordForQueryChar(MatchContext* ctx,
                              unsigned int qryid,
                              unsigned int qrychar) {
  MatchResults* results = &(ctx->results);
  MatchCoord* coords = results->h_match_coords;
#if COALESCED_QUERIES
  return coords + results->h_coord_tex_array[qryid] + qrychar * WARP_SIZE;
#else
  return coords + results->h_coord_tex_array[qryid] + qrychar;
#endif
}

void coordsToPrintBuffers(MatchContext* ctx,
                          ReferencePage* page,
                          MatchInfo** matches,
                          Alignment** alignments,
                          unsigned int mem_avail,
                          unsigned int* coord_idx,
                          unsigned int* match_idx,
                          unsigned int* align_idx,
                          unsigned int* nextqry,
                          unsigned int* nextqrychar) {
  unsigned int numQueries = ctx->queries->count;
  int match_length = ctx->min_match_length;
  unsigned int cidx = *coord_idx;
  unsigned int midx = 0;

  unsigned int numCoords = ctx->results.numCoords;

  unsigned int numMatches = 0;
  unsigned int numAlignments = 0;

  int DEBUG = 0;
  if (DEBUG && cidx == 0) {
    for (int j = 0; j < numCoords; ++j) {
      MatchCoord * coord = ctx->results.h_match_coords+j;
      if (coord->node.data > 0 && !(coord->edge_match_length & FRMASK)) {
        //fprintf(stdout, "node: %d\n",
        //        coord->node);
        fprintf(stdout, "node: %d leaves:%d\n",
                coord->node.data, lookupNumLeaves(page, coord->node));
      }
    }
    exit(0);
  }


  // How much can we fit into mem_avail?
  for (int j = cidx; j < numCoords; ++j) {
    MatchCoord* coord = ctx->results.h_match_coords + j;

    int queryAlignments = 0;
    int queryMatches = 0;

    if (coord->node.data > 0 && !(coord->edge_match_length & FRMASK)) {
      int numLeaves = lookupNumLeaves(page, coord->node);
      queryAlignments += numLeaves;
      queryMatches++;
    }
    int allMatches    = numMatches    + queryMatches;
    int allAlignments = numAlignments + queryAlignments;

    int neededSize = allMatches * sizeof(MatchInfo) + allAlignments * sizeof(Alignment);

    if (neededSize > mem_avail || (allMatches/BLOCKSIZE) >= MAX_GRID_DIMENSION) {
      // adding this match won't fit on the board
      break;
    }

    ++cidx;
    numMatches    = allMatches;
    numAlignments = allAlignments;
  }

  MatchInfo* M = (MatchInfo*)calloc(numMatches, sizeof(MatchInfo));
  unsigned int alignmentOffset = 0;

  int qry = *nextqry;
  int qrychar = *nextqrychar;
  bool set_full = false;
  while (qry < numQueries) {
    // h_lengths_array doesn't count the 'q' at the beginning of each query
    int qlen = ctx->queries->h_lengths_array[qry] + 1 - match_length;

    while (qrychar < qlen) {
      if (midx >= numMatches) {
        set_full = true;
        break;
      }

      MatchCoord* coord = coordForQueryChar(ctx, qry, qrychar);

      if (coord->node.data > 0 && !(coord->edge_match_length & FRMASK)) {
        MatchInfo m;
        m.resultsoffset = alignmentOffset;
        m.qrystartpos = qrychar;
        m.matchnode = coord->node;
        m.edgematch = coord->edge_match_length;
        m.numLeaves = lookupNumLeaves(page, m.matchnode);
        m.queryid = qry;

        alignmentOffset += m.numLeaves;
        M[midx++] = m;
      }

      ++qrychar;
    }

    if (set_full)
      break;

    ++qry;
    qrychar = 0;
  }

  *coord_idx = cidx;
  *match_idx = midx;
  *align_idx = alignmentOffset;
  *matches = M;
  *nextqry = qry;
  *nextqrychar = qrychar;
  fprintf(stderr, "Allocing %ld bytes of host memory for %d alignments\n",
          alignmentOffset * sizeof(Alignment), numAlignments);
  *alignments = (struct Alignment *) calloc(alignmentOffset, sizeof(Alignment));
  //hipHostMalloc((void**)alignments, numAlignments * sizeof(Alignment));
}


// TODO: need reverse-complement printing support
void runPrintOnCPU(MatchContext* ctx, ReferencePage* page,
                   MatchInfo* h_matches,
                   unsigned int numMatches,
                   Alignment* alignments,
                   unsigned int numAlignments) {
  unsigned int min_match_length = ctx->min_match_length;

  int* addrs = ctx->queries->h_addrs_tex_array;
  int* lengths = ctx->queries->h_lengths_array;
  char* qrychars = ctx->queries->h_tex_array;

  if (!numMatches)
    return;

  int qry = -1;
  unsigned int qrylen;

  for (int i = 0; i < numMatches; ++i) {
    MatchInfo& match = h_matches[i];
    if (match.queryid != qry) {
      qry = match.queryid;
      qrylen = lengths[qry];
    }
    if (!(match.edgematch & FRMASK)) {
      printAlignments(page,
                      alignments + match.resultsoffset,
#if COALESCED_QUERIES
                      qrychars + sizeof(int) * addrs[qry],
#else
                      qrychars + addrs[qry],
#endif
                      qrylen,
                      match.matchnode,
                      match.qrystartpos,
                      match.edgematch,
                      min_match_length,
                      0,
                      ctx->forwardcoordinates);
    }
  }
}

int addMatchToBuffer(int left_in_ref, int qrypos, int matchlen);

void getExactAlignments(MatchContext * ctx, ReferencePage * page, bool on_cpu) {
  assert(!ctx->reverse && !ctx->forwardreverse);

  size_t boardFreeMemory;

  if (!on_cpu) {
  } else {
    boardFreeMemory = 256 * 1024 * 1024;
  }

#ifdef __DEVICE_EMULATION__
  boardFreeMemory = 512 * 1024 * 1024;
#endif

  boardFreeMemory -= BREATHING_ROOM;
  fprintf(stderr, "board free memory: %lu\n", boardFreeMemory);

  int rTotalMatches = 0;
  int rTotalAlignments = 0;
  int totalRounds = 0;
  unsigned int last_coord = ctx->results.numCoords;
  unsigned int next_coord = 0;
  unsigned int nextqry = 0;
  unsigned int nextqrychar = 0;
  int lastqry = -1;
  while (next_coord < last_coord) {
    // see how many queries will fit on the board
    totalRounds++;

    unsigned int numMatches = 0;
    unsigned int numAlignments = 0;
    MatchInfo* h_matches = NULL;
    Alignment* h_alignments = NULL;
    int coord_left = next_coord;
    char* btimer = createTimer();
    startTimer(btimer);
    coordsToPrintBuffers(ctx, page, &h_matches, &h_alignments, boardFreeMemory,
                         &next_coord, &numMatches, &numAlignments, &nextqry, &nextqrychar);
    stopTimer(btimer);

    float btime = getTimerValue(btimer);
    ctx->statistics.t_coords_to_buffers += btime;
    fprintf(stderr, "buffer prep time= %f\n", btime);
    deleteTimer(btimer);

    fprintf(stderr, "Round %d: Printing results for match coords [%d-%d) of %d using %d matches and %d alignments\n",
            totalRounds, coord_left, next_coord, last_coord, numMatches, numAlignments);

    if (numMatches == 0)
      continue;

    char buf[256];
    //assert(qryend > qrystart);

    rTotalAlignments += numAlignments;
    rTotalMatches += numMatches;

    char* ktimer = createTimer();
    startTimer(ktimer);
    if (on_cpu) {
      runPrintOnCPU(ctx, page, h_matches, numMatches,
                    h_alignments, numAlignments);
    } else {
    }
    stopTimer(ktimer);

    float ktime = getTimerValue(ktimer);
    ctx->statistics.t_print_kernel += ktime;
    fprintf(stderr, "print kernel time= %f\n", ktime);
    deleteTimer(ktimer);

    // char* stimer = createTimer();
    // startTimer(stimer);
    // mapQueriesEndToEnd(ctx,
    //                    page,
    //                    h_matches,
    //                    numMatches,
    //                    h_alignments,
    // 				   numAlignments);
    //
    // stopTimer(stimer);
    //
    // float stime = getTimerValue(stimer);
    // fprintf(stderr, "postprocess time= %f\n", stime);
    // deleteTimer(stimer);

    //flushOutput();

    //Process the alignments
    char* otimer = createTimer();
    startTimer(otimer);

    for (int m = 0; m < numMatches; m++) {
      int base = h_matches[m].resultsoffset;
      for (int i = 0; i < h_matches[m].numLeaves; i++) {
        // See if there are any more left maximal alignments for this match
        if (h_alignments[base+i].left_in_ref == 0) {
          break;
        }

        if (h_matches[m].queryid != lastqry) {
          lastqry = h_matches[m].queryid;
          addToBuffer("> ");
          addToBuffer(*(ctx->queries->h_names + lastqry));
          addToBuffer("\n");
        }

        sprintf(buf, "%d\t%d\t%d\n",
                h_alignments[base+i].left_in_ref,
                h_matches[m].qrystartpos + 1,
                h_alignments[base+i].matchlen);
        addToBuffer(buf);

        // addMatchToBuffer(h_alignments[base+i].left_in_ref,
        // 								 h_matches[m].qrystartpos + 1,
        // 								h_alignments[base+i].matchlen);

      }
    }


    flushOutput();

    stopTimer(otimer);
    ctx->statistics.t_results_to_disk += getTimerValue(otimer);
    deleteTimer(otimer);

    free(h_matches);
    free(h_alignments);
    //hipHostFree((void*)h_alignments);

  }
  free(ctx->results.h_coord_tex_array);
  //hipHostFree(ctx->results.h_match_coords);
  free(ctx->results.h_match_coords);
  ctx->results.h_coord_tex_array = NULL;
  ctx->results.h_match_coords = NULL;

  fprintf(stderr, "Finished processing %d matches and %d potential alignments in %d rounds\n",
          rTotalMatches, rTotalAlignments, totalRounds);
}

int getQueryBlock(MatchContext* ctx, size_t device_mem_avail) {
  QuerySet* queries = ctx->queries;
  char * queryTex = NULL;
  int* queryAddrs = NULL;
  int* queryLengths = NULL;
  unsigned int numQueries;
  unsigned int num_match_coords;
  size_t queryLen;
  char** names;

  fprintf(stderr, "Loading query block... ");

  char* queryreadtimer = createTimer();
  startTimer(queryreadtimer);

  getQueriesTexture(queries->qfile,
                    &queryTex,
                    &queryLen,
                    &queryAddrs,
                    &names,
                    &queryLengths,
                    &numQueries,
                    &num_match_coords,
                    device_mem_avail,
                    ctx->min_match_length,
                    ctx->reverse || ctx->forwardreverse);

  stopTimer(queryreadtimer);
  ctx->statistics.t_queries_from_disk += getTimerValue(queryreadtimer);
  deleteTimer(queryreadtimer);

  queries->h_tex_array = queryTex;
  queries->count = numQueries;
  queries->h_addrs_tex_array = queryAddrs;
  queries->texlen = queryLen;
  queries->h_names = names;
  queries->h_lengths_array = queryLengths;

  ctx->results.numCoords = num_match_coords;

  fprintf(stderr, "done.\n");

  return numQueries;
}

void destroyQueryBlock(QuerySet* queries) {
  free(queries->h_tex_array);
  queries->h_tex_array = NULL;

  for (int i = 0; i < queries->count; ++i)
    free(queries->h_names[i]);

  free(queries->h_names);

  queries->count = 0;
  queries->texlen = 0;

  free(queries->h_addrs_tex_array);
  queries->h_addrs_tex_array = NULL;

  free(queries->h_lengths_array);
  queries->h_lengths_array = NULL;
}

void resetStats(Statistics* stats) {
  stats->t_end_to_end = 0.0;
  stats->t_match_kernel = 0.0;
  stats->t_print_kernel = 0.0;
  stats->t_queries_to_board = 0.0;
  stats->t_match_coords_to_board = 0.0;
  stats->t_match_coords_from_board = 0.0;
  stats->t_tree_to_board = 0.0;
  stats->t_ref_str_to_board = 0.0;
  stats->t_queries_from_disk = 0.0;
  stats->t_ref_from_disk = 0.0;
  stats->t_results_to_disk = 0.0;
  stats->t_tree_construction = 0.0;
  stats->t_tree_reorder = 0.0;
  stats->t_tree_flatten = 0.0;
  stats->t_reorder_ref_str = 0.0;
  stats->t_build_coord_offsets = 0.0;
  stats->t_coords_to_buffers = 0.0;
  stats->bp_avg_query_length = 0.0;

#if TREE_ACCESS_HISTOGRAM
  if (stats->node_hist_size) {
    free(stats->node_hist);
    stats->node_hist = NULL;
    stats->node_hist_size = 0;
  }

  if (stats->child_hist_size) {
    free(stats->child_hist);
    stats->child_hist = NULL;
    stats->child_hist_size = 0;
  }
#endif
}

void writeStatisticsFile(Statistics* stats,
                         char* stats_filename,
                         char* node_hist_filename = NULL,
                         char* child_hist_filename = NULL) {
  if (stats_filename) {
    FILE* f = fopen(stats_filename, "w");

    if (!f) {
      fprintf(stderr, "WARNING: could not open %s for writing\n", stats_filename);
    } else {
      fprintf(f, "Q");
      fprintf(f, ",R");
      fprintf(f, ",T");
      fprintf(f, ",m");
      fprintf(f, ",r");
      fprintf(f, ",t");
      fprintf(f, ",n");
      fprintf(f, ",Total");
      fprintf(f, ",Match kernel");
      fprintf(f, ",Print Kernel");
      fprintf(f, ",Queries to board");
      fprintf(f, ",Match coords to board");
      fprintf(f, ",Match coords from board");
      fprintf(f, ",Tree to board");
      fprintf(f, ",Ref str to board");
      fprintf(f, ",Queries from disk");
      fprintf(f, ",Ref from disk");
      fprintf(f, ",Output to disk");
      fprintf(f, ",Tree construction");
      fprintf(f, ",Tree reorder");
      fprintf(f, ",Tree flatten");
      fprintf(f, ",Ref reorder");
      fprintf(f, ",Build coord table");
      fprintf(f, ",Coords to buffers");
      fprintf(f, ",Avg qry length");
      fprintf(f, "\n");

      fprintf(f, "%d", QRYTEX);
      fprintf(f, ",%d", REFTEX);
      fprintf(f, ",%d", TREETEX);
      fprintf(f, ",%d", MERGETEX);
      fprintf(f, ",%d", REORDER_REF);
      fprintf(f, ",%d", REORDER_TREE);
      fprintf(f, ",%d", RENUMBER_TREE);
      fprintf(f, ",%f", stats->t_end_to_end);
      fprintf(f, ",%f", stats->t_match_kernel);
      fprintf(f, ",%f", stats->t_print_kernel);
      fprintf(f, ",%f", stats->t_queries_to_board);
      fprintf(f, ",%f", stats->t_match_coords_to_board);
      fprintf(f, ",%f", stats->t_match_coords_from_board);
      fprintf(f, ",%f", stats->t_tree_to_board);
      fprintf(f, ",%f", stats->t_ref_str_to_board);
      fprintf(f, ",%f", stats->t_queries_from_disk);
      fprintf(f, ",%f", stats->t_ref_from_disk);
      fprintf(f, ",%f", stats->t_results_to_disk);
      fprintf(f, ",%f", stats->t_tree_construction);
      fprintf(f, ",%f", stats->t_tree_reorder);
      fprintf(f, ",%f", stats->t_tree_flatten);
      fprintf(f, ",%f", stats->t_reorder_ref_str);
      fprintf(f, ",%f", stats->t_build_coord_offsets);
      fprintf(f, ",%f", stats->t_coords_to_buffers);
      fprintf(f, ",%f", stats->bp_avg_query_length);
      fprintf(f,"\n");

      fclose(f);
    }
  }
#if TREE_ACCESS_HISTOGRAM
  if (node_hist_filename) {
    FILE* f = fopen(node_hist_filename, "w");
    if (!f) {
      fprintf(stderr, "WARNING: could not open %s for writing\n", node_hist_filename);
    } else {
      for (unsigned int i = 0; i < ctx->statistics.node_hist_size; ++i)
        fprintf(f, "%d\t%d\n", i, ctx->statistics.node_hist[i]);
    }

  }

  if (child_hist_filename) {
    FILE* f = fopen(child_hist_filename, "w");
    if (!f) {
      fprintf(stderr, "WARNING: could not open %s for writing\n", child_hist_filename);
    } else {
      for (unsigned int i = 0; i < ctx->statistics.child_hist_size; ++i)
        fprintf(f, "%d\t%d\n", i, ctx->statistics.child_hist[i]);
    }

  }

  float total_node_hits = 0;
  float tree_top_node_hits = 0;

  float total_child_hits = 0;
  float tree_top_child_hits = 0;

  for (unsigned int i = 0; i < ctx->statistics.node_hist_size; ++i) {
    total_node_hits +=ctx->statistics.node_hist[i];
    if (i < 256) {
      tree_top_node_hits += ctx->statistics.node_hist[i];
    }
  }

  for (unsigned int i = 0; i < ctx->statistics.child_hist_size; ++i) {
    total_child_hits +=ctx->statistics.child_hist[i];
    if (i < 256) {
      tree_top_child_hits += ctx->statistics.child_hist[i];
    }
  }

  fprintf(stderr, "Tree top node  hits (%d/%d) = %f percent\n",(int)tree_top_node_hits, (int)total_node_hits, tree_top_node_hits /total_node_hits);
  fprintf(stderr, "Tree top child hits (%d/%d) = %f percent\n",(int)tree_top_child_hits, (int)total_child_hits, tree_top_child_hits /total_child_hits);
#endif
}

void matchOnCPU(MatchContext* ctx, bool doRC) {
  //TODO: CPU is matching is disabled.
  if (doRC) {
    // Match the reverse complement of the queries to the ref
    computeGold(&ctx->results,
                ctx->ref->str,
                ctx->queries->h_tex_array,
                ctx->queries->h_addrs_tex_array,
                ctx->queries->h_lengths_array,
                (PixelOfNode*)(ctx->ref->h_node_tex_array),
                (PixelOfChildren*)(ctx->ref->h_children_tex_array),
                ctx->queries->count,
                ctx->min_match_length,
                REVERSE);
  } else {
    computeGold(&ctx->results,
                ctx->ref->str,
                ctx->queries->h_tex_array,
                ctx->queries->h_addrs_tex_array,
                ctx->queries->h_lengths_array,
                (PixelOfNode*)(ctx->ref->h_node_tex_array),
                (PixelOfChildren*)(ctx->ref->h_children_tex_array),
                ctx->queries->count,
                ctx->min_match_length,
                FORWARD);
  }
}


void matchQueryBlockToReferencePage(MatchContext* ctx,
                                    ReferencePage* page,
                                    bool reverse_complement) {
  char*  ktimer = createTimer();

  fprintf(stderr, "Memory footprint is:\n\tqueries: %ld\n\tref: %ld\n\tresults: %ld\n",
          ctx->queries->bytes_on_board,
          ctx->ref->bytes_on_board,
          ctx->results.bytes_on_board);

  startTimer(ktimer);
  if (ctx->on_cpu) {
    matchOnCPU(ctx, reverse_complement);
  } else {

  }
  stopTimer(ktimer);

  float ktime = getTimerValue(ktimer);
  ctx->statistics.t_match_kernel += ktime;
  fprintf(stderr, "match kernel time= %f\n", ktime);
  deleteTimer(ktimer);

}


int matchSubset(MatchContext* ctx,
                ReferencePage* page) {

  loadQueries(ctx);

  fprintf(stderr,
          "Matching queries %s - %s against ref coords %d - %d\n",
          ctx->queries->h_names[0],
          ctx->queries->h_names[ctx->queries->count - 1],
          page->begin,
          page->end);

  loadResultBuffer(ctx);

  // TODO: renable RC support by calling this twice /w reverse/fwdreverse
  // idiom.
  matchQueryBlockToReferencePage(ctx, page, false);

  if (USE_PRINT_KERNEL && !ctx->on_cpu) {
  } else {
    getExactAlignments(ctx, page, true);
  }

  flushOutput();
  return 0;
}

int getFreeDeviceMemory(bool on_cpu) {
  size_t free_mem = 0;

  // We have to 'prime' CUDA by making an allocation here.  hipMemGetInfo
  // will return zeroes until we do a malloc.
  if (!on_cpu) {

  } else {
    free_mem = 804585472; // pretend we are on a 8800 GTX
  }

  return free_mem;
}

int matchQueriesToReferencePage(MatchContext* ctx, ReferencePage* page) {
  fprintf(stderr, "Beginning reference page %p\n", page);

  int free_mem = getFreeDeviceMemory(ctx->on_cpu);

  int available_mem = free_mem - page->ref.bytes_on_board - BREATHING_ROOM;
  ctx->ref = &(page->ref);
  loadReference(ctx);

  while (getQueryBlock(ctx, available_mem)) {
    matchSubset(ctx, page);
    ctx->statistics.bp_avg_query_length =
      ctx->queries->texlen / (float)(ctx->queries->count) - 2;
    destroyQueryBlock(ctx->queries);
  }

  unloadReferenceString(ctx->ref);
  lseek(ctx->queries->qfile, 0, SEEK_SET);
  return 0;
}



void initReferencePages( MatchContext* ctx , int* num_pages, ReferencePage** pages_out) {
  unsigned int bases_in_ref = ctx->full_ref_len - 3;
  unsigned int page_size = BASES_PER_TREE_PAGE < bases_in_ref ?
                           BASES_PER_TREE_PAGE : bases_in_ref;
  unsigned int num_reference_pages = ceil((bases_in_ref + 0.0) / page_size);
  fprintf(stderr, "Stream will use %d pages for %d bases, page size = %d\n",
          num_reference_pages, bases_in_ref, page_size);

  unsigned int page_overlap = MAX_QUERY_LEN + 1;
  ReferencePage* pages = (ReferencePage*) calloc(num_reference_pages,
                         sizeof(ReferencePage));

  pages[0].begin = 1;
  pages[0].end = pages[0].begin +
                 page_size  +
                 ceil(page_overlap / 2.0) + 1; //the 1 is for the 's' at the beginning
  pages[0].shadow_left = -1;
  pages[0].id = 0;

  for (int i = 1; i < num_reference_pages - 1; ++i) {
    pages[i].begin = pages[i - 1].end - page_overlap;
    pages[i].end = pages[i].begin + page_size +  page_overlap;
    pages[i - 1].shadow_right = pages[i].begin;
    pages[i].shadow_left = pages[i-1].end;
    pages[i].id = i;
  }

  if (num_reference_pages > 1) {
    int last_page = num_reference_pages - 1;
    pages[last_page].begin = pages[last_page - 1].end - page_overlap;
    pages[last_page].end = ctx->full_ref_len - 1;
    pages[last_page - 1].shadow_right = pages[last_page].begin;
    pages[last_page].shadow_right = -1;
    pages[last_page].shadow_left = pages[last_page - 1].end;
    pages[last_page].id = last_page;
  }

  *pages_out = pages;
  *num_pages = num_reference_pages;
}

int streamReferenceAgainstQueries(MatchContext* ctx) {
  int num_reference_pages = 0;
  ReferencePage* pages = NULL;
  initReferencePages(ctx, &num_reference_pages, &pages);


  buildReferenceTexture(&(pages[0].ref),
                        ctx->full_ref,
                        pages[0].begin,
                        pages[0].end,
                        ctx->min_match_length,
                        ctx->dotfilename,
                        ctx->texfilename,
                        &(ctx->statistics));


  matchQueriesToReferencePage(ctx, &pages[0]);
  destroyReference(&(pages[0].ref));

  for (int i = 1; i < num_reference_pages - 1; ++i) {

    buildReferenceTexture(&(pages[i].ref),
                          ctx->full_ref,
                          pages[i].begin,
                          pages[i].end,
                          ctx->min_match_length,
                          NULL,
                          NULL,
                          &(ctx->statistics));

    matchQueriesToReferencePage(ctx, &pages[i]);
    destroyReference(&(pages[i].ref));
  }

  if (num_reference_pages > 1) {
    int last_page = num_reference_pages - 1;
    buildReferenceTexture(&(pages[last_page].ref),
                          ctx->full_ref,
                          pages[last_page].begin,
                          pages[last_page].end,
                          ctx->min_match_length,
                          NULL,
                          NULL,
                          &(ctx->statistics));

    matchQueriesToReferencePage(ctx, &pages[last_page]);
    destroyReference(&(pages[last_page].ref));
  }
  free(pages);
  return 0;
}


extern "C"
int matchQueries(MatchContext* ctx) {
  assert(sizeof(struct PixelOfNode) == sizeof(uint4));
  assert(sizeof(struct PixelOfChildren) == sizeof(uint4));

#if TREE_ACCESS_HISTOGRAM
  ctx->statistics.node_hist_size = 0;
  ctx->statistics.child_hist_size = 0;
#endif

  resetStats(&(ctx->statistics));

  char* ttimer = createTimer();
  startTimer(ttimer);

  int ret;

  fprintf(stderr, "Streaming reference pages against all queries\n");
  ret = streamReferenceAgainstQueries(ctx);

  stopTimer(ttimer);
  ctx->statistics.t_end_to_end += getTimerValue(ttimer);
  deleteTimer(ttimer);

  writeStatisticsFile(&(ctx->statistics), ctx->stats_file, "node_hist.out", "child_hist.out");

  return ret;
}



